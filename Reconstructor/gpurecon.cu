
#include "gpurecon.h"

#ifdef __cplusplus
extern "C"
{
#endif
   
///Pointer to the reconstructor on the GPU, allocated and populated by calling init_gpurecon
float * rec_gpu;

///Pointer to memory for slopes on the GPU, allocated by calling init_gpurecon, updated with each call to gpurecon
float * slopes_gpu;

///Pointer to the amplitude vector on the GPU, allocated by calling init_gpurecon, filled with result of sgemv
float * amps_gpu;

///Number of modes in reconstructor (matrix rows, length or amp vector)
int n_modes;

///Number of slopes in reconstructor (matrix columns, length of slopes vector)
int n_slopes;

///Used bye cublas library
hipblasHandle_t handle;

///Get the current time as a double.
double get_curr_t()
{
   struct timespec tsp;
   clock_gettime(CLOCK_REALTIME, &tsp);
   
   return ((double)tsp.tv_sec) + ((double)tsp.tv_nsec)/1e9;
}


int init_gpurecon(int nm, int ns, float *rec_host)
{

   hipError_t cudaStat;
   hipblasStatus_t stat;

   //Initialize the cublas library
   stat = hipblasCreate(&handle);
   if ( stat != HIPBLAS_STATUS_SUCCESS )
   {
      fprintf (stderr, "CUBLAS initialization failed\n" ) ;
      return EXIT_FAILURE;
   }
   
   n_modes = nm;
   n_slopes = ns;

   //Convert to column major storage.
   float *colmaj = (float *) malloc(nm*ns*sizeof(float));

   if(colmaj == 0)
   {
      fprintf(stderr, "Allocation of main memory for reconstructor col-major failed.\n");
      return EXIT_FAILURE;
   }
   
   for(int i=0;i<nm;i++)
   {
      for(int j=0;j<ns;j++)
      {
         colmaj[j*nm +i] = rec_host[i*ns + j];
      }
   }
   
   cudaStat = hipMalloc((void **)&rec_gpu, n_modes*n_slopes*sizeof(float));
   if( cudaStat != hipSuccess )
   {
      fprintf(stderr, "GPU memory allocation failed\n") ;
      return EXIT_FAILURE ;
   }
   
   cudaStat = hipMalloc((void **)&slopes_gpu, n_slopes * sizeof(float));
   if( cudaStat != hipSuccess )
   {
      fprintf(stderr, "GPU memory allocation failed\n") ;
      return EXIT_FAILURE ;
   }
   
   cudaStat = hipMalloc((void **) &amps_gpu, n_modes*sizeof(float));
   if( cudaStat != hipSuccess )
   {
      fprintf(stderr, "GPU memory allocation failed\n") ;
      return EXIT_FAILURE ;
   }   
   //stat = hipblasSetMatrix ( M , N , s i z e o f (* a ) , a , M , devPtrA , M ) ;
   
   stat = hipblasSetMatrix(n_modes, n_slopes, sizeof(float), colmaj, n_modes, rec_gpu, n_modes);
   
   if ( stat != HIPBLAS_STATUS_SUCCESS)
   {
      fprintf(stderr, "Error loading reconstructor onto GPU\n") ;
      return EXIT_FAILURE ;
   }

   free(colmaj);
   
   return EXIT_SUCCESS;

}

int free_gpurecon()
{
   hipFree(rec_gpu);
   hipFree(slopes_gpu);
   hipFree(amps_gpu);

   hipblasDestroy(handle);

   return 0;
}


int gpurecon(float *slopes_host, float *amps_host)
{
   hipblasStatus_t stat;
   
   float alpha = 1.0f, beta = 0.0f;
   
   //double t0, t1, t2;
   
   //t0 = get_curr_t();
   
   stat = hipblasSetVector(n_slopes, sizeof(float), slopes_host, 1, slopes_gpu, 1);
   if(stat != HIPBLAS_STATUS_SUCCESS)
   {
      fprintf(stderr, "Error sending slopes vector to GPU.\n");
      return EXIT_FAILURE;
   }
   //t1 = get_curr_t();
   
   stat = hipblasSgemv(handle, HIPBLAS_OP_N, n_modes, n_slopes, &alpha, rec_gpu, n_modes, slopes_gpu, 1, &beta, amps_gpu, 1);
   if(stat != HIPBLAS_STATUS_SUCCESS)
   {
      fprintf(stderr, "Error during matrix-vector multiply.\n");
      return EXIT_FAILURE;
   }
   //t2 = get_curr_t();

   stat = hipblasGetVector(n_modes, sizeof(float), amps_gpu, 1, amps_host, 1);
   if(stat != HIPBLAS_STATUS_SUCCESS)
   {
      fprintf(stderr, "Error getting amplitudes vector from GPU.\n");
      return EXIT_FAILURE;
   }
   /*if(tw)
   {
      *tr = get_curr_t() - t2;
      *tmm = t2 - t1;
      *tw = t1 - t0;
   }*/
   
   return EXIT_SUCCESS;

   
}

#ifdef __cplusplus
}//extern "C"
#endif

